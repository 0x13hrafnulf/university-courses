/*******************************************************************************
*
*   This is the code from Chapter 3 of CUDA by Example illustrating how
*   vector addition is accomplished by the GPU.
*
*   compile with:
*     > nvcc vectorAdd_gpu.cu
*
*******************************************************************************/

#include <hip/hip_runtime.h>
#include <stdio.h>

/******************************************************************************/
// KERNEL TO RUN ON THE GPU:
__global__ void addGPU(int *a, int *b, int *c, int vecSize)
{
  int tid = blockIdx.x;         // get thread ID (tid) from the kernel
  if (tid < vecSize){           // only process memory that has been allocated
    c[tid] = a[tid] + b[tid];   // do addition based on memory location
  }
}

/******************************************************************************/
int main(void)
{

  int vecSize = 10; // set size of vectors to add (play with this)
  int intSize = sizeof(int); // get size in bytes of an int on this system

  // allocate memory on CPU (host) for three arrays
  int a[vecSize], b[vecSize], c[vecSize];

  // fill the host memory with numbers
  for (int i = 0; i < vecSize; i++) {
      a[i] = -i;
      b[i] = i * i;
      c[i] = 0;
  }

  // print what is stored in host memory
  printf("before GPU processing:\n");
  for (int i=0; i<vecSize; i++) {
    printf( "%d + %d = %d\n", a[i], b[i], c[i] );
  }

  // allocate memory on the GPU (device)
  int *dev_a, *dev_b, *dev_c;
  hipMalloc((void**)&dev_a, intSize*vecSize);
  hipMalloc((void**)&dev_b, intSize*vecSize);
  hipMalloc((void**)&dev_c, intSize*vecSize);

  // copy vector a and vector b from host (CPU) memory to device (GPU) memory
  hipMemcpy(dev_a, a, intSize*vecSize, hipMemcpyHostToDevice);
  hipMemcpy(dev_b, b, intSize*vecSize, hipMemcpyHostToDevice);

  // launch needed number of threads on the GPU, each doing vector addition
  int numBlocks = vecSize;
  int threadsPerBlock = 1;
  addGPU <<< numBlocks, threadsPerBlock >>> (dev_a, dev_b, dev_c, vecSize);

  // copy thre result from device (GPU) memory to host (CPU) memory
  hipMemcpy(c, dev_c, intSize*vecSize, hipMemcpyDeviceToHost);

  // be a good citizen and release allocated memory on the device (GPU)
  hipFree(dev_a);
  hipFree(dev_b);
  hipFree(dev_c);

  // print what is now stored in host memory
  printf("\nafter GPU processing:\n");
  for (int i=0; i<vecSize; i++) {
    printf( "%d + %d = %d\n", a[i], b[i], c[i] );
  }

return 0;
}

/******************************************************************************/
