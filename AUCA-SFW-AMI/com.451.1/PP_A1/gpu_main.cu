#include "hip/hip_runtime.h"
/**************************************************************************
*
*  NOTES ABOUT PROGRAM GO HERE
*
**************************************************************************/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include "gpu_main.h"

/*************************************************************************/
void FuncGPU(int *a, float *b, unsigned long vecSize){
  int intSize = sizeof(int);
  int floatSize = sizeof(float);

  int *dev_a;
  float *dev_b;
  hipMalloc((void**)&dev_a, intSize*vecSize);
  hipMalloc((void**)&dev_b, floatSize*vecSize);

  // copy vector a and vector b from host (CPU) memory to device (GPU) memory
  hipMemcpy(dev_a, a, intSize*vecSize, hipMemcpyHostToDevice);
  hipMemcpy(dev_b, b, floatSize*vecSize, hipMemcpyHostToDevice);
  // launch needed number of threads on the GPU, each doing vector addition
  int numBlocks = vecSize;
  int threadsPerBlock = 1;
  func <<< numBlocks, threadsPerBlock >>> (dev_a, dev_b, vecSize);

  // copy thre result from device (GPU) memory to host (CPU) memory
  hipMemcpy(b, dev_b, floatSize*vecSize, hipMemcpyDeviceToHost);

  // be a good citizen and release allocated memory on the device (GPU)
  hipFree(dev_a);
  hipFree(dev_b);

  // print what is now stored in host memory
}
__global__ void func(int *a, float *b, unsigned long vecSize)
{
	int tid = blockIdx.x;
	for(int N = 1; N <= a[tid]; ++N)
	{
		if(N % 2 == 0) b[tid] *= N;
		else b[tid] /= N;
	}
}

void GPUinfo()
{
	int deviceCount = 0;
	hipGetDeviceCount(&deviceCount);
	if(deviceCount == 0)
	{
		printf("There are no available device(s) supporting CUDA");
	}
	else
	{
		printf("Detected %d CUDA capable device(s)\n", deviceCount);
	}
	for(int dev = 0; dev < deviceCount; ++dev){

		int driverVersion = 0, runtimeVersion = 0;

		hipSetDevice(dev);
		hipDeviceProp_t deviceProp;
		hipGetDeviceProperties(&deviceProp, dev);
		printf("Device %d: \"%s\"\n", dev, deviceProp.name);

		hipDriverGetVersion(&driverVersion);
		hipRuntimeGetVersion(&runtimeVersion);
		printf(" CUDA Driver Version / Runtime Version %d.%d / %d.%d\n",
			driverVersion/1000, (driverVersion%100)/10,
			runtimeVersion/1000, (runtimeVersion%100)/10);
		printf(" CUDA Capability Major/Minor version number: %d.%d\n",
			deviceProp.major, deviceProp.minor);
		printf(" Total amount of global memory: %.2f MBytes (%llu bytes)\n",
			(float)deviceProp.totalGlobalMem/(pow(1024.0,3)),
			(unsigned long long) deviceProp.totalGlobalMem);
		printf(" GPU Clock rate: %.0f MHz (%0.2f GHz)\n",
			deviceProp.clockRate * 1e-3f, deviceProp.clockRate * 1e-6f);
		printf(" Memory Clock rate: %.0f Mhz\n",
			deviceProp.memoryClockRate * 1e-3f);
		printf(" Memory Bus Width: %d-bit\n",
			deviceProp.memoryBusWidth);
		if (deviceProp.l2CacheSize) {
			printf(" L2 Cache Size: %d bytes\n", deviceProp.l2CacheSize);
		}
		printf(" Max Texture Dimension Size (x,y,z)		"
			"	1D=(%d), 2D=(%d,%d), 3D=(%d,%d,%d)\n",
			deviceProp.maxTexture1D, deviceProp.maxTexture2D[0],
		deviceProp.maxTexture2D[1], deviceProp.maxTexture3D[0],
       		deviceProp.maxTexture3D[1], deviceProp.maxTexture3D[2]);

		printf(" Max Layered Texture Size (dim) x layers	1D=(%d) x %d, 2D=(%d,%d) x %d\n",
			deviceProp.maxTexture1DLayered[0], deviceProp.maxTexture1DLayered[1],
			deviceProp.maxTexture2DLayered[0], deviceProp.maxTexture2DLayered[1],
			deviceProp.maxTexture2DLayered[2]);

		printf(" Total amount of constant memory:	%lu bytes\n",
				deviceProp.totalConstMem);
		printf(" Total amount of shared memory per block: 	%lu bytes\n",
				deviceProp.sharedMemPerBlock);
		printf(" Total number of registers available per block:		%d\n",
			       	deviceProp.regsPerBlock);
		printf(" Warp size:	%d\n", deviceProp.warpSize);
		printf(" Maximum number of threads per multiprocessor:	%d\n",
		deviceProp.maxThreadsPerMultiProcessor);
		printf(" Maximum number of threads per block:	%d\n",
			deviceProp.maxThreadsPerBlock);
		printf(" Maximum sizes of each dimension of a block:	%d x %d x %d\n",
			deviceProp.maxThreadsDim[0],
			deviceProp.maxThreadsDim[1],
			deviceProp.maxThreadsDim[2]);
		printf(" Maximum sizes of each dimension of a grid:	%d x %d x %d\n",
			deviceProp.maxGridSize[0],
			deviceProp.maxGridSize[1],
			deviceProp.maxGridSize[2]);
		printf(" Maximum memory pitch:	%lu bytes\n", deviceProp.memPitch);
		printf("\n");
	}
}
	/*************************************************************************/
