#include "hip/hip_runtime.h"
/**************************************************************************
*
*
*
**************************************************************************/

#include <hip/hip_runtime.h>

#include <stdio.h>
#include "gpu_main.h"
#include "params.h"
#include <math.h>

texture<float,2> texture_blue;

/*************************************************************************/
__global__ void updateReds(float* red, hiprandState* states)
{
  int x = threadIdx.x + (blockIdx.x * blockDim.x);
  int y = threadIdx.y + (blockIdx.y * blockDim.y);
  int vecIdx = x + (y * blockDim.x * gridDim.x);

  hiprandState randState = states[vecIdx];
  float rndNumber = hiprand_uniform(&randState);
  states[vecIdx] = randState;

  red[vecIdx] = (red[vecIdx] + rndNumber) / 2.0f;
}
__global__ void setup_random(hiprandState* states, long seed)
{
  int x = threadIdx.x + (blockIdx.x * blockDim.x);
  int y = threadIdx.y + (blockIdx.y * blockDim.y);
  int vecIdx = x + (y * blockDim.x * gridDim.x);

  hiprand_init(seed, vecIdx, 0, &states[vecIdx]);
}
/*************************************************************************/
__global__ void updateGreens(float* green, float sinV)
{
  int x = threadIdx.x + (blockIdx.x * blockDim.x);
  int y = threadIdx.y + (blockIdx.y * blockDim.y);
  int vecIdx = x + (y * blockDim.x * gridDim.x);


  green[vecIdx] = sinV;//(PI/180) * time --- mb use that one for test
}
/*************************************************************************/
__global__ void updateBlues(float* blue)//neighbors means 10x10 area? or just vertical and horizontal ones
{
  int x = threadIdx.x + (blockIdx.x * blockDim.x);
  int y = threadIdx.y + (blockIdx.y * blockDim.y);
  int vecIdx = x + (y * blockDim.x * gridDim.x);

  /*float sum = tex2D(texture_blue, x, y);;
  int count = 1;
  for(int i = 1; i <= 5; ++i)
  {
    sum += tex2D(texture_blue, x, y - i);//top
    sum += tex2D(texture_blue, x, y + i);//bottom
    sum += tex2D(texture_blue, x - i, y);//left
    sum += tex2D(texture_blue, x + i, y);//right
    //diagonals
    sum += tex2D(texture_blue, x - i, y - i);//
    sum += tex2D(texture_blue, x + i, y + i);//
    sum += tex2D(texture_blue, x - i, y + i);//
    sum += tex2D(texture_blue, x + i, y - i);//
    count += 8;
  }
  */
  float sum = 0.0f;
  for(int i = -5; i <= 5; ++i)
  {
    for(int j = -5; j <= 5; ++j)
    {
      if(i == 0 && j == 0) continue;
      sum += tex2D(texture_blue, x + i, y + i);
    }
  }

  blue[vecIdx] = sum / 120;
}

/*************************************************************************/
int updatePalette(GPU_Palette* P, int mode, float t){

//  do something <<< P->gBlocks, P->gThreads >>> (P->red);
  if(mode == 1)
  {
      //copy_const_kernel <<< P->gBlocks, P->gThreads >>> (P->blue);

      updateBlues <<< P->gBlocks, P->gThreads >>> (P->blue);
  }
  else if(mode == 2)
  {

    updateReds <<< P->gBlocks, P->gThreads >>> (P->red, P->theRand);
  }
  else if(mode == 3)
  {
      float sinValue = sin(t);
      updateGreens <<< P->gBlocks, P->gThreads >>> (P->green, sinValue);
  }
  return 0;
}

/*************************************************************************/
GPU_Palette initGPUPalette(AParams* PARAMS){

  // load
  GPU_Palette P;

  P.gTPB = 32;      // threads per block
  //P.gDIM = ((PARAMS->height) * (PARAMS->width)) / (32 * 32);     // assumes the image is 800x800
  P.gDIMy = PARAMS->height;
  P.gDIMx = PARAMS->width;
  // 800x800 palette = 25x25 grid of 32x32 threadblocks
  P.gSize = (P.gDIMy * P.gDIMx) * sizeof(float);// Area * float size
  long gRandSize = (P.gDIMy * P.gDIMx) * sizeof(hiprandState);
  P.gThreads.x = P.gTPB;
  P.gThreads.y = P.gTPB;
  P.gThreads.z = 1;         // 3D of threads allowed
  P.gBlocks.x = P.gDIMx/P.gTPB;
  P.gBlocks.y = P.gDIMy/P.gTPB;
  P.gBlocks.z = 1;          // only 2D of blocks allowed

  // allocate memory for the palette
  hipMalloc((void**) &P.gray, P.gSize);    // black and white (avg of rgb)
  hipMalloc((void**) &P.red, P.gSize);   // r
  hipMalloc((void**) &P.green, P.gSize); // g
  hipMalloc((void**) &P.blue, P.gSize);  // b

  //task 1 creating const data point to update based on time step
  hipChannelFormatDesc desc = hipCreateChannelDesc<float>();
  hipBindTexture2D(NULL, texture_blue, P.blue, desc, P.gDIMx, P.gDIMy, sizeof(float) * P.gDIMx);
  //task 2, update red = avg(red + randomValueOfThread)
  hipMalloc((void**) &P.theRand, gRandSize);
  setup_random <<<P.gBlocks, P.gThreads>>> (P.theRand, time(NULL));
  //task 3, green = sin(t)

  return P;
}


/*************************************************************************/
int freeGPUPalette(GPU_Palette* P) {

  hipUnbindTexture(texture_blue);
  hipFree(P->blue);
  hipFree(P->red);
  hipFree(P->green);
  hipFree(P->gray);
  hipFree(P->theRand);
  return 0;
}

/*************************************************************************/
